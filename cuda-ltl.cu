#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-ltl.cu - Larger than Life with CUDA using shared memory
 *
 * Written by Martina Cavallucci <martina.cavallucci(at)studio.unibo.it>
 * --------------------------------------------------------------------------
 *
 * This version of the Larger than Life uses shared memory and should
 * work correctly with any domain size n.
 * Compile with:
 *
 * nvcc cuda-ltl.cu -o cuda-ltl
 *
 * Run with:
 * (Test it with the Game of Life parameter )
 * ./cuda-ltl (R =) 1 (B1 = B2 =) 3 (D1 =) 3  (D2 = )4 nsteps input_file output_file
 *
 ****************************************************************************/
 #include "hpc.h"
 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include <assert.h>
 #include <ctype.h> /* for isdigit */

#define BLKSIZE 32
/*we assume the presence of 8 rows / columns of ghost cells per side,
 then using only those that serve*/
#define HALO 8
/* We use 1D blocks to copy ghost cells; in this case we can use up to
   1024 threads per block (for the GPUs on the lab machine) */

#define BLKSIZE_GHOST 1024

 typedef unsigned char cell_t;

 /* The struct of the bmap_t with a size and a point of unsigned char */

 typedef struct {
     int n;
     cell_t *bmap;
 } bmap_t;

 /* The following function makes indexing of the two-dimensional CA
    grid easier. Instead of writing, e.g., grid[i][j] (which you can
    not do anyway, since the CA grids are passed around as pointers to
    linear blocks of data), you write IDX(grid, n, i, j) to get a
    pointer to grid[i][j]. This function assumes that the size of the
    CA grid is (n+2)*(n+2), where the first and last rows/columns are
    ghost cells.

    Note the use of both the __device__ and __host__ qualifiers: this
    function can be called both from host and device code. */

 __device__ __host__ cell_t *IDX(cell_t *grid, int n, int i, int j)
 {
 	return (grid + i*(n +2*HALO)+j);
 }

/* Fill the ghost cells of |grid| in order to have cyclic boundary conditions*/
__global__ void copy_top_bottom(cell_t *grid, int n)
{

  const int end = HALO + n - 1;
  const int j = HALO + threadIdx.x + blockIdx.x * blockDim.x;
  const int k =  threadIdx.y + blockIdx.y * blockDim.y + 1;
  /* Copy top and bottom */

  if( k < HALO + 1){
    if ( j < end + 1) {
      *IDX(grid, n, end + k, j) = *IDX(grid, n, HALO + k - 1, j);
      *IDX(grid, n, HALO - k, j) = *IDX(grid, n, end - k + 1, j);
      }
    }
}
__global__ void copy_left_right(cell_t *grid, int n )
{

  const int end = HALO + n - 1;
  const int i = HALO + threadIdx.y + blockIdx.y * blockDim.y;
  const int k =  threadIdx.y + blockIdx.y * blockDim.y + 1;
  /* Copy left and right */

  if( k < HALO + 1){
    if ( i < end + 1 ) {
      *IDX(grid, n, i, end + k) = *IDX(grid, n, i, HALO + k - 1);
      *IDX(grid, n, i, HALO - k) = *IDX(grid, n, i, end + k - 1);
    }
  }
}
__global__ void copy_corners(cell_t *grid, int n)
{
    const int i =  threadIdx.y + blockIdx.y * blockDim.y;
    const int j = threadIdx.x + blockIdx.x * blockDim.x;

    /* Copy corners*/
    if( i < HALO){
      if (j < HALO){
        *IDX(grid, n, i ,j) = *IDX(grid, n, i + HALO + 1, j + HALO + 1 );
        *IDX(grid, n, i + HALO + n ,j + HALO + n) = *IDX(grid, n, i + HALO, j + HALO);
        *IDX(grid, n, i ,j + HALO + n) = *IDX(grid, n, i + HALO + 1, j + HALO );
        *IDX(grid, n, i + HALO + n ,j) = *IDX(grid, n, i + HALO , j + HALO + 1 );
      }
  }
}
/**
  * Write the content of the bmap_t structure pointed to by ltl to the
  * file f in PBM format and and allocates space
  *for the ghost cell that will be assigned.
  * The caller is responsible for passing a
  * pointer f to a file opened for writing
*/
 void write_ltl( bmap_t* grid, FILE *f , int r )
 {
     const int n = grid->n;
     fprintf(f, "P1\n");
     fprintf(f, "# produced by ltl\n");
     fprintf(f, "%d %d\n", n, n);
     for (int i = r ; i < n + r; i++) {
        for (int j = r ; j < n + r; j++) {
             fprintf(f, "%d ", *IDX(grid->bmap, n, i, j));
         }
         fprintf(f, "\n");
     }
 }

 /*Compute of the Larger than life*/
__global__ void compute_ltl( cell_t *cur, cell_t *next, int n, int r, int b1, int b2, int d1, int d2)
 {
   /*we assume the presence of 8 rows / columns of ghost cells per side,
    then using only those that serve*/

   __shared__ cell_t buf[BLKSIZE+2*HALO][BLKSIZE+2*HALO];

   /* "global" indexes */
   const int gi = HALO + threadIdx.y + blockIdx.y * blockDim.y;
   const int gj = HALO + threadIdx.x + blockIdx.x * blockDim.x;
   /* "local" indexes */
   const int li = HALO + threadIdx.y;
   const int lj = HALO + threadIdx.x;
   int nbors = 0;

    /*Copy elements from global memory to local memory of block*/
    if ( gi<n+2*HALO && gj<n+2*HALO ) {
        buf[li][lj] = *IDX(cur, n, gi, gj);

        if (li < 2*HALO) { /* left-right */
            buf[li-HALO   ][lj] = *IDX(cur, n, gi-HALO, gj);
            buf[li+BLKSIZE][lj] = (gi+BLKSIZE < n+2*HALO ? *IDX(cur, n, gi+BLKSIZE, gj) : 0);
        }
        if (lj < 2*HALO) { /* top-bottom */
            buf[li][lj-HALO   ] = *IDX(cur, n, gi, gj-HALO);
            buf[li][lj+BLKSIZE] = (gj+BLKSIZE < n+2*HALO ? *IDX(cur, n, gi, gj+BLKSIZE) : 0);
        }
        if (li < 2*HALO && lj < 2*HALO) { /* corners */
          buf[li-HALO   ][lj-HALO   ] = *IDX(cur, n, gi-HALO, gj-HALO);
          buf[li-HALO   ][lj+BLKSIZE] = (gj+BLKSIZE < n+2*HALO ? *IDX(cur, n, gi-HALO, gj+BLKSIZE) : 0);
          buf[li+BLKSIZE][lj-HALO   ] = (gi+BLKSIZE < n+2*HALO ? *IDX(cur, n, gi+BLKSIZE, gj-HALO) : 0);
          buf[li+BLKSIZE][lj+BLKSIZE] = (gi+BLKSIZE < n+2*HALO && gj+BLKSIZE < n+2*HALO ? *IDX(cur, n, gi+BLKSIZE, gj+BLKSIZE) : 0);
        }
    }
    __syncthreads();

    const int globali = r + threadIdx.y + blockIdx.y * blockDim.y;
    const int globalj = r + threadIdx.x + blockIdx.x * blockDim.x;
    const int localy = r + threadIdx.y;
    const int localx = r + threadIdx.x;
    int i,j;
    for(i = localy - r ; i < localy + r ; i++){
      for(j = localx - r; j < localx + r; j++){
          nbors = nbors +
          buf[i][j] ;
        }
    }

    if( !buf[localx][localy] && nbors >= b1 && nbors <= b2){ // if it can relive
      *IDX(next, n, globali, globalj) = 1; //Set it as live

    }else if(buf[localx][localy] && nbors + 1 >= d1 && nbors + 1 <= d2) // if the cell remaining live
    {
      *IDX(next, n, globali, globalj) = 1;// set it as live
    }else{
      *IDX(next, n, globali, globalj) = 0; // set it as died
    }
}



 /**
  * Read a PBM file from file f. The caller is responsible for passing
  * a pointer f to a file opened for reading. This function is not very
  * robust; it may fail on perfectly legal PBM images, but should work
  * for the images produced by gen-input.c. Also, it should work with
  * PBM images produced by Gimp (you must save them in "ASCII format"
  * when prompted).
  */
 void read_ltl( bmap_t *ltl, FILE* f, int r)
 {
    char buf[2048];
    char *s;
    int n, i, j;
    int width, height;

     /* Get the file type (must be "P1") */
    s = fgets(buf, sizeof(buf), f);
    if (0 != strcmp(s, "P1\n")) {
        fprintf(stderr, "FATAL: Unsupported file type \"%s\"\n", buf);
        exit(-1);
    }
    /* Get any comment and ignore it; does not work if there are
       leading spaces in the comment line */
    do {
        s = fgets(buf, sizeof(buf), f);
    } while (s[0] == '#');
    /* Get width, height; since we are assuming square images, we
       reject the input if width != height. */
    sscanf(s, "%d %d", &width, &height);
    if ( width != height ) {
        fprintf(stderr, "FATAL: image width (%d) and height (%d) must be equal\n", width, height);
        exit(-1);
    }
    ltl->n = n = width;
    int ng = n + (2*HALO);
    ltl->bmap = (cell_t*)malloc( ng * ng * sizeof(cell_t));
    /* scan bitmap; each pixel is represented by a single numeric
       character ('0' or '1'); spaces and other separators are ignored
       (Gimp produces PBM files with no spaces between digits) */
    for (i = HALO; i < n + HALO ; i++) {
         for (j = HALO; j < n + HALO ; j++) {
            int val;
            do {
                val = fgetc(f);
                if ( EOF == val ) {
                    fprintf(stderr, "FATAL: error reading input\n");
                    exit(-1);
                }
            } while ( !isdigit(val) );
            *IDX(ltl->bmap, n, i, j) = (val - '0');
        }

    }
 }

 int main( int argc, char* argv[] )
 {
     int R, B1, B2, D1, D2, nsteps,s;
     const char *infile, *outfile;
     FILE *in, *out;
     bmap_t cur;
     cell_t *d_cur, *d_next,*d_tmp;
     double tstart, tend;
     if ( argc != 9 ) {
         fprintf(stderr, "Usage: %s R B1 B2 D1 D2 nsteps infile outfile\n", argv[0]);
         return -1;
     }
     R = atoi(argv[1]);
     B1 = atoi(argv[2]);
     B2 = atoi(argv[3]);
     D1 = atoi(argv[4]);
     D2 = atoi(argv[5]);
     nsteps = atoi(argv[6]);
     infile = argv[7];
     outfile = argv[8];

     assert(  R <= 8  );
     assert(  0 <= B1 );
     assert( B1 <= B2 );
     assert(  1 <= D1 );
     assert( D1 <= D2 );

     in = fopen(infile, "r");
     if (in == NULL) {
         fprintf(stderr, "FATAL: can not open \"%s\" for reading\n", infile);
         exit(-1);
     }
     read_ltl(&cur, in, R);
     fclose(in);

     fprintf(stderr, "Size of input image: %d x %d\n", cur.n, cur.n);
     fprintf(stderr, "Model parameters: R=%d B1=%d B2=%d D1=%d D2=%d nsteps=%d\n",
             R, B1, B2, D1, D2, nsteps);

     out = fopen(outfile, "w");
     if ( out == NULL ) {
         fprintf(stderr, "FATAL: can not open \"%s\" for writing", outfile);
         exit(-1);
     }

     const int n = cur.n;
     dim3 cpyBlock(BLKSIZE_GHOST,HALO);
     /*------DA SISTEMARE NOMI ----*/
     /*there was 1 al posto di HALO nel secondo parametro*/
     //dim3 cpyGrid(( n + 2 *HALO + BLKSIZE_GHOST - 1) / BLKSIZE_GHOST,HALO);
     dim3 cpyGrid((n + BLKSIZE-1)/BLKSIZE, (n + BLKSIZE-1)/BLKSIZE);
     dim3 stepBlock(BLKSIZE,BLKSIZE);
     dim3 stepGrid((n + BLKSIZE-1)/BLKSIZE, (n + BLKSIZE-1)/BLKSIZE);

     const size_t size = (n+2*HALO)*(n+2*HALO)*sizeof(*(cur.bmap));
    /* Allocate space for device copy of cur and next grids */
    hipMalloc((void**)&d_cur, size);
    hipMalloc((void**)&d_next, size);

    /* Copy initial grid to d_cur */
	  hipMemcpy(d_cur,cur.bmap, size, hipMemcpyHostToDevice);
    tstart = hpc_gettime();

    for (s = 0; s < nsteps; s++) {
       copy_top_bottom<<<stepGrid,stepBlock>>>(d_cur, n);
       copy_left_right<<<stepGrid,stepBlock>>>(d_cur, n);
       copy_corners<<<stepGrid,stepBlock>>>(d_cur, n);
       compute_ltl<<<stepGrid,stepBlock>>>(d_cur, d_next, n, R, B1, B2, D1, D2);
       d_tmp = d_cur;
       d_cur = d_next;
       d_next = d_tmp;
    }
    hipDeviceSynchronize();
    tend = hpc_gettime();
    fprintf(stderr, "Execution time %f\n", tend - tstart);
    hipMemcpy(cur.bmap, d_cur, size, hipMemcpyDeviceToHost);
    write_ltl(&cur, out, R);
    fclose(out);
    free(cur.bmap);
    hipFree(d_cur);
    hipFree(d_next);
     return 0;
 }
